#define MAX_INPUTS 17

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hiprand/hiprand_kernel.h"


__device__ static char tlds[37][3] = {
    "ar", "at", "au", "cc", "ch", "cn", "co", "de", "dk", "es", "eu", "fm", "fr", "gr", "hk", "hu", "io", "is", "it", "kr", "ly", "me", "mx", "nl", "no", "nz", "pt", "pw", "rs", "ru", "se", "sg", "si", "tn", "tw", "uk", "us"
};


// Returns the product of the two given field elements modulo GF(2^8/0x11D).
// All inputs are valid. This could be implemented as a 256*256 lookup table.
__device__ __attribute__((always_inline)) inline unsigned char reedSolomonMultiply(unsigned char x, unsigned char y) {
	// Russian peasant multiplication
	unsigned char z = 0;
	for (int i = 7; i >= 0; i--) {
		z = (unsigned char)((z << 1) ^ ((z >> 7) * 0x11D));
		z ^= ((y >> i) & 1) * x;
	}
	return z;
}

template<int degree>
__device__ __attribute__((noinline)) void fill_out_table(unsigned char *table) {
    unsigned char generator[degree] = {127, 122, 154, 164, 11, 68, 117};
    #pragma unroll 1
    for (int i = 0; i < 256; i++) {
        #pragma unroll 7 // otherwise generator goes in local memory
        for (int j = 0; j < 7; j++) {
            table[i*8 + j] = reedSolomonMultiply(generator[j], i);
        }
        table[i*8+7] = 0;
    }
}

template<int degree, int len>
__device__ __attribute__((always_inline)) inline unsigned int calculate_bitcount(unsigned long long *ll_table, unsigned char input_data[17]) {
    constexpr int dataLen = 19;
    unsigned char new_data[dataLen] = {0};
    new_data[0] = 0b01000000 | ((len&0b11110000)>>4);
    new_data[1] = ((len&0b1111)<<4);

    for (int i = 0; i < len; i++) {
        new_data[i+1] |= ((input_data[i]&0b11110000)>>4);
        new_data[i+2] |= ((input_data[i]&0b1111)<<4);
    }

    // ideally this is precomputed
    bool EC = true; // alternating EC and 11
    for (int i = len; i < MAX_INPUTS; i++) {
        unsigned char val = EC ? 0xEC : 0x11;
        EC = !EC;
        new_data[i+2] = val;
    }

    unsigned char generator[7] = {127, 122, 154, 164, 11, 68, 117};

    int offset = 0;
    unsigned char ecc_data[7] = {0};
    unsigned long long ecc_data_ll = 0;
    #pragma unroll dataLen
	for (int i = 0; i < dataLen; i++) {  // Polynomial division
        int zero_idx = (degree + offset - 1)%degree;
        unsigned char factor = new_data[i] ^ ((ecc_data_ll & (0xFFULL<<(zero_idx*8)))>>(zero_idx*8));
        ecc_data_ll &= (0xFFFFFFFFFFFFFFFFULL^(0xFFULL << (zero_idx*8)));
        // ecc_data[zero_idx] = 0;
        unsigned long long table_val = ll_table[factor];
        // printf("factor is %02X, table_val is %llu, ecc_data_ll is %llu\n", factor, table_val, ecc_data_ll);
        // printf("ecc_data_ll is %llu\n", ecc_data_ll);
        #pragma unroll degree
		for (int j = 0; j < degree; j++) {
            unsigned int v = (j+offset);
            unsigned int idx = (v)%degree;
            unsigned long long table_ll = (table_val & (0xFFULL<<(j*8)))>>(j*8);
            ecc_data_ll ^= (table_ll << (idx*8));
            // ecc_data[idx] ^= reedSolomonMultiply(generator[j], factor);
        }
        offset = (offset+1)%degree;
    }
    /*
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("For input data \"%.17s\", ECC:\n", input_data);
        for (int i = 0; i < 8; i++) {
            printf("ecc: %02X\t", ecc_data[i]);
            unsigned int result = (ecc_data_ll & (0xFFULL<<(i*8)))>>(i*8);
            printf("ll %02X\n", result);
        }
    }
    */

    unsigned char all_codewords[26];
    memcpy(all_codewords, new_data, 19);

    for (int i = 0; i < 7; i++) {
        int idx = (i+4)%7;
        all_codewords[19+i] = (ecc_data_ll & (0xFFULL<<(idx*8)))>>(idx*8);
    }

    unsigned char masks[8][26] = {
        { 153,153,153,102,102,102,153,153,153,102,102,102,153,153,153,150,102,153,150,102,102,102,153,153,102,153 },
        { 204,204,204,51,51,51,204,204,204,51,51,51,204,204,204,195,51,204,195,51,51,51,204,51,204,51 },
        { 0,0,0,170,170,170,85,85,85,0,0,0,170,170,170,170,170,85,85,85,85,85,0,0,170,85 },
        { 97,134,24,146,73,36,134,24,97,36,146,73,24,97,134,17,134,73,34,73,36,146,24,36,134,73 },
        { 195,195,195,150,150,150,60,60,60,195,195,195,150,150,150,154,90,15,12,60,60,60,195,60,105,195 },
        { 12,32,194,171,170,186,76,100,198,131,8,48,174,170,234,170,171,198,65,147,25,49,194,67,201,87 },
        { 124,167,202,191,171,250,206,108,230,163,218,61,175,234,254,175,171,206,99,155,57,179,202,111,205,87 },
        { 11,208,189,98,118,39,185,27,145,126,7,224,216,157,137,210,118,185,30,70,228,110,189,129,118,185 }
    };

    int max_count = 0;
    int min_mask = 20;
    #pragma unroll 8
    for (int mask = 5; mask < 7; mask++) {
        int total_count = 0;
        #pragma unroll 26
        for (int i = 0; i < 26; i++) {
            total_count += __builtin_popcount(all_codewords[i] ^ masks[mask][i]);
        }
        // lots of black or lots of white
        total_count = abs(104-total_count);
        if (total_count > max_count) {
            max_count = total_count;
            min_mask = mask;
        }
    }

    return (max_count << 16) + min_mask;
}

/*
__device__ static inline char random_char(curandState_t &state) {
    char val = (curand(state) % 62)+48;
    if (val <= '9') {
        return val;
    }
    val += 8; // 9 -> A
    if (val <= 'A') {
        return val;
    }
    val += 7;
    return val;
}
*/

#define RANDOM_CHAR(state) ( \
    { \
        char val = (hiprand(&state) % 62) + 48; \
        (val <= '9') ? val : ((val+8) <= 'Z') ? val+8 : val+13; \
    } \
)

__global__ void generate_qrs(int *lock, unsigned char *output) {
    unsigned int seed = blockIdx.x * gridDim.x + threadIdx.x + lock[64];
    hiprandState_t state;
    hiprand_init(seed, 0, 0, &state);

    unsigned char input_data[17];

    input_data[4] = '.';

    input_data[7] = '/';

    constexpr int len = 17;

    constexpr int degree = 7;
    __shared__ unsigned long long ll_table[256];
    fill_out_table<degree>((unsigned char *)ll_table);

    unsigned long long final_result = 0;
    #pragma unroll 9
    for (int iter = 0; iter < (1000*1000*1000); iter++) {
        if ((iter % (10*1000)) == 0) {
            input_data[0] = RANDOM_CHAR(state);
            input_data[1] = RANDOM_CHAR(state);
            input_data[2] = RANDOM_CHAR(state);
            input_data[3] = RANDOM_CHAR(state);

            int tld_idx = hiprand(&state) % 37;
            input_data[5] = tlds[tld_idx][0];
            input_data[6] = tlds[tld_idx][1];
        }
        if (iter%676 == 0) {
            #pragma unroll 9
            for (int i = 0; i < 9; i++) {
                input_data[i+8] = RANDOM_CHAR(state); 
            }
        }
        input_data[iter%9 + 8] = RANDOM_CHAR(state);

        unsigned int ret = calculate_bitcount<degree, len>(ll_table, input_data);
        // printf("Got value %d for string %.17s\n", count, input_data);
        if (ret >= (46 << 16)) {
            unsigned char mask = (ret & 0xFFFF);
            unsigned char count = ((ret&0xFFFF0000)>>16);
            int output_index = atomicAdd(lock, 1);
            #pragma unroll 17
            for (int input_idx = 0; input_idx < 17; input_idx++) {
                output[(output_index*19)+input_idx] = input_data[input_idx];
            }
            output[(output_index*19)+17] = mask + '0';
            output[(output_index*19)+18] = count;
        }
    }
}

int run_qr(int *lock, unsigned char *output) {
    int blocks = 256;
    int threads = 256;
    generate_qrs<<<blocks, threads>>>(lock, output);
    // generate_qrs<<<1, 1>>>(lock, output);
    return 0;
}